
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
using namespace std;
//max no. of blocks 65535

__global__ void square(long long *d_in,long long *d_out,long long d_n,long long *d_get_blockDim)
{
	long long idx=1024*blockIdx.x+threadIdx.x;
	if(idx<d_n)
	{
		int temp=d_in[idx];
		d_out[idx]=temp*temp;
		d_out[idx]=idx;
	}
	*d_get_blockDim=blockDim.x;
//	cout<<threadIdx.x<<" "<<b[threadIdx.x]<<" ";
}

int main()
{
	long long n;
	cin>>n;
	long long *h_a;
	h_a=(long long *)malloc(n*sizeof(n));
	long long *h_b;
	h_b=(long long *)malloc(n*sizeof(long long));
	for(int i=0;i<n;i++)
		h_a[i]=i;
	long long *d_get_blockDim;
	
	long long *d_in;
	long long *d_out;
	long long d_n=n;
	hipMalloc((void **)&d_in,n*sizeof(n));
	hipMalloc((void **)&d_out,n*sizeof(long long));
	hipMalloc((void **)&d_get_blockDim,sizeof(long long));
	
	hipMemcpy(d_in,h_a,n*sizeof(n),hipMemcpyHostToDevice);
	
	clock_t start,end;
	
	start=clock();
//	long long i=0;
	for(long long i=0;i<n;i+=67107840)
	{
		long long noBlocks=(n-i)/1024;
		if(noBlocks<(n-i)/1024.0)
			noBlocks++;
		square <<<noBlocks,1024>>> (d_in+i,d_out+i,d_n-i,d_get_blockDim);
	}

	end=clock();
	
	hipMemcpy(h_b,d_out,n*sizeof(n),hipMemcpyDeviceToHost);
/*	
	for(int i=0;i<n;i++)
		cout<<h_b[i]<<" ";
*/
	long long h_blockDim;
	hipMemcpy(&h_blockDim,d_get_blockDim,sizeof(long long),hipMemcpyDeviceToHost);
	cout<<h_b[n-1]<<"\n";
	cout<<"Block Dimension is:"<<h_blockDim<<endl;
	
	free(h_a);
	free(h_b);
	hipFree(d_in);
	hipFree(d_out);
	printf("It took %0.9f seconds\n",(end-start)*1.0/CLOCKS_PER_SEC);
}
